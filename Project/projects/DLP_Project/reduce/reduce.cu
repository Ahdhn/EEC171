#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#define NUM_BANKS 16
#define N_ELEMENTS 16384

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>

// includes, kernels
__global__ void sum_kernel(float *g_odata, float *g_idata, int n)
{
    int i;

    int tid = threadIdx.x; //Calculate a thread ID based on this thread's position within the block
    //int tid = threadIdx.y * blockDim.x + threadIdx.x; //Another thread ID example for a 2-D thread block
    //int tid = blockIdk.x * blockDim.x + threadIdx.x; //Another thread ID example for assigning unique thread IDs across
    //different blocks

    g_odata[0] = 0;

    //A single thread adds up all 1M array elements serially.
    //This is a poor use of parallel hardware - your job is to increase the number of threads, split up the work, and communicate
    //data between threads as necessary to improve the kernel performance.
    for(i = 0;i < N_ELEMENTS;i++)
    {
        g_odata[0] += g_idata[i];
    }

    __syncthreads(); //Syncthreads forces all threads within a block to reach this point before continuing past. Note this is
    //necessary within blocks because not all threads can physically execute at the same time.
    //Syncthreads does NOT synchronize different blocks (but you should not need to for this project).
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                      const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    hipSetDevice( cutGetMaxGflopsDeviceId() );

    int num_elements = N_ELEMENTS;
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", &num_elements);

    unsigned int timer;
    cutilCheckError( cutCreateTimer(&timer));
    
    const unsigned int num_threads = 1;
    const unsigned int mem_size = sizeof( float) * num_elements;

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host to be integer values
    // between 0 and 1000
    printf("INPUT: ");
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
        printf(" %f ", h_data[i]);
    }
    printf("\n");

    // compute reference solution
    float* reference = (float*) malloc( mem_size);  
    computeGold( reference, h_data, num_elements);

    // allocate device memory input and output arrays
    float* d_idata;
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device input array
    cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );

    // setup execution parameters
    // Note that these scans only support a single thread-block worth of data,
    // but we invoke them here on many blocks so that we can accurately compare
    // performance
#ifndef __DEVICE_EMULATION__
    dim3  grid(1, 1, 1);  
#else
    dim3  grid(1, 1, 1); // only one run block in device emu mode or it will be too slow
#endif
    dim3  threads(num_threads, 1, 1);

    // make sure there are no CUDA errors before we start
    CUT_CHECK_ERROR("Kernel execution failed");

    printf("Running sum of %d elements\n", num_elements);
  
    // execute the kernels
    unsigned int numIterations = 100;

    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        sum_kernel<<< grid, threads >>>
            (d_odata, d_idata, num_elements);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);

    cutResetTimer(timer);

    // check for any errors
    cutilCheckMsg("Kernel execution failed");

    // check results
    // copy result from device to host
    cutilSafeCall(hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                                   hipMemcpyDeviceToHost));

    printf("OUTPUT: ");
    printf(" %f ", h_data[0]);
    printf("\n");
    printf("REFERENCE: ");
    printf(" %f ", reference[0]);
    printf("\n");

    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected soluion
    
    // Due to the large number of additions, a non-zero epsilon is necessary to
    // mask floating point precision errors.
    float epsilon = 0.0f;
    unsigned int result_regtest = cutComparefe( reference, h_data, 1, epsilon);
    printf( "sum: Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");

    // cleanup memory
    free( h_data);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));
    cutilCheckError(cutDeleteTimer(timer));
}
